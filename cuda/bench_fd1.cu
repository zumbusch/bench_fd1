#include "hip/hip_runtime.h"
/* -*- mode: c++ -*-  */

// Copyright (c) 2011, 2012, 2014, Gerhard Zumbusch
// All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:

// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.

// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.

// * The names of its contributors may not be used to endorse or promote
//   products derived from this software without specific prior written
//   permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.


// 1D FD, 3pt stencil, periodic b.c.
// space-time slicing

// change parameters and look for output flop=...
// optimize the kernel "diagvuu"

// size of local O(memory/(2*sizeof(real))
// tuning parameter, larger is better, fill device memory


// number of GPUs
#ifndef DEV_MAX
#define DEV_MAX 1
#endif

// check for small iteration numbers
// #define CHECK
// #define PRINT

// number of (multi-) processors on a GPU 
#ifndef PROC
#define PROC 5
#endif

#include "gpu_cuda.hpp"
#include <stdlib.h>
#include "stdio.h"

using namespace std;

//#define REAL2
//#define WRP 32

#ifdef FLOAT
typedef float real;
#else
typedef double real;
#endif


// algorithm: even number of time steps
#ifndef TIMESTEP
#define TIMESTEP 50
#endif

// algorithm: number of vectors width
#ifndef WIDTH
#define WIDTH 50
#endif

// number of threads on a GPU = algorithm vector length
#ifndef LOCAL
#define LOCAL 256
#endif

// large LOCAL and large WIDTH exceed GPU register limit

// algorithm: approx grid size 
#ifndef GRIDSIZE
#define GRIDSIZE 314572800/8
#endif


// round to even multiple of LOCAL * WIDTH * PROC
#define GRID_LOCAL (((GRIDSIZE) + (LOCAL) * (WIDTH) * (PROC) - 1) / (2 * (LOCAL) * (WIDTH) * (PROC)) * 2)


#if (GRID_LOCAL < 2 * TIMESTEP)
#error "overlap too large"
#endif

#if ((WIDTH) <= 1)
#error "WIDTH must be greater 1"
#endif

#if ((LOCAL)%32 != 0)
#warning "LOCAL should be a multiple of 32"
#endif

#if ((TIMESTEP)%2 != 0)
#error "TIMESTEP must be even"
#endif

#if ((GRID_LOCAL)%2 != 0)
#warning "even GRID_LOCAL"
#endif


// ----------------------------------------------------------------------
// initial data
// ----------------------------------------------------------------------

extern "C" __global__ void kernel0 (real *gx, uint m, uint dev_no) {
  int i = threadIdx.x;
  int k0 = blockIdx.x * LOCAL*WIDTH*GRID_LOCAL;
  int o = 2*TIMESTEP*LOCAL;
  for (int n=0; n<GRID_LOCAL; n++)
    for (int ii=0; ii<WIDTH; ii++) {
      real y = ((m * dev_no / LOCAL + blockIdx.x *WIDTH*GRID_LOCAL+ n*WIDTH + ii) + i*((m*DEV_MAX)/LOCAL)) / (real) (m * DEV_MAX);
      y = y*y;
      if (DEV_MAX!=1) {
  	int j = k0 + n*WIDTH*LOCAL + ii*LOCAL + i;
  	gx[j] = y;
      } else {
  	for (int j1=0; j1<2+o/m; j1++) {
  	  int j = j1*m + k0 + n*WIDTH*LOCAL + ii*LOCAL + (i+j1)%LOCAL;
  	  if (j<o+m) {
  	    gx[j] = y;
	  }
  	}
      }
    }
}


// ----------------------------------------------------------------------
// rotate boundary condition vectors
// ----------------------------------------------------------------------

extern "C" __global__ void kernel1 (real *gx, uint m) {
  int i = threadIdx.x;
  int k0 = blockIdx.x *LOCAL + m;
  gx[k0 + i] = gx[k0 + (i+LOCAL-1)%LOCAL];
}

// ----------------------------------------------------------------------

__device__  __host__ inline real kern (real a, real b, real c) {
  // 3pt stencil
  real d = .5f * b + .25f * (a + c);
  return d;
}


//----------------------------------------------------------------------
// alternative versions of kernel2
//----------------------------------------------------------------------

#ifndef WRP
#ifdef REAL2
//----------------------------------------------------------------------

#ifdef FLOAT
typedef float2 real2;
#else
typedef double2 real2;
#endif

__device__ void diagvu2 (real2 *ap, real2 *bp, uint m) {
  // 2*unrolled space-time slice
  int id = threadIdx.x;
  real2 dd = bp [0+id];
  real d0 = dd.x;
  real d1 = dd.y;
  for (int i=0; i<m*LOCAL; i+=LOCAL) {
    real2 ad = ap [i+id];
    real a0 = ad.x;
    real a1 = ad.y;
    real e0 = kern (a0, a1, d0);
    real e1 = kern (a1, d0, d1);
    bp [i+LOCAL+id] = (real2){e0, e1};
    d0 = e0;
    d1 = e1;
  }
}

__device__ void diagvuu (real2 *ap, real2 *bp, real *ip, real *jp) {
  // u*unrolled in space, 2*in time, space-time slice
  int id = threadIdx.x;
  ap[id] = (real2){ip [id], ip [LOCAL+id]};
  real d[WIDTH+2];
  for (int k=0; k<WIDTH; k++)
    d[k+2] = ip [LOCAL*(k+2)+id];
  for (int i=0; i<TIMESTEP*LOCAL; i+=2*LOCAL) {
    real2 dd = ap [i+id];
    d[0] = dd.x;
    d[1] = dd.y;
    real e[WIDTH+2];
    for (int k=0; k<WIDTH; k++)
      e[k+2] = kern (d[k], d[k+1], d[k+2]);
    bp [i+LOCAL+id] = (real2){e[WIDTH], e[WIDTH+1]};
    real2 ed = ap [i+LOCAL+id];
    e[0] = ed.x;
    e[1] = ed.y;
    for (int k=0; k<WIDTH; k++)
      d[k+2] = kern (e[k], e[k+1], e[k+2]);
    bp [i+2*LOCAL+id] = (real2){d[WIDTH], d[WIDTH+1]};
  }
  for (int k=0; k<WIDTH; k++)
    jp [LOCAL*k+id] = d[k+2];
}


extern "C" __global__ void kernel2 (real *s0, real *s1,
				    real *ga, real *gb) {
  int id = threadIdx.x;
  int k1 = blockIdx.x *LOCAL*2*(TIMESTEP+1);

  real2 *a = (real2*)&ga[k1];
  real2 *b = (real2*)&gb[k1];
  int i0 = blockIdx.x *LOCAL*GRID_LOCAL*WIDTH;
  int i1 = (blockIdx.x+1) *LOCAL*GRID_LOCAL*WIDTH;
  a [0+id] = (real2){s0 [i0+id], s0 [i0+LOCAL+id]};
  for (int i=1; i<TIMESTEP; i++) { // 2* unroll initial
    b [0+id] = (real2){s0 [2*LOCAL*i+i0+id], s0 [2*LOCAL*i+LOCAL+i0+id]};
    diagvu2 (a, b, i);
    real2 *c = a;
    a = b;
    b = c;
  }

  // if (GRID_LOCAL%2 == 1) {
  //   for (int i=i0; i<i1; i+=LOCAL*(WIDTH)) { // u* unroll block
  //     real *ip = &s0 [i+2*LOCAL*(TIMESTEP-1)];
  //     real *jp = &s1 [i];
  //     diagvuu (a, b, ip, jp);
  //     real *c = a;
  //     a = b;
  //     b = c;
  //   }
  // } else {
    for (int i=i0; i<i1; i+=2*LOCAL*(WIDTH)) { // u* unroll block
      real *ip = &s0 [i+2*LOCAL*(TIMESTEP-1)];
      real *jp = &s1 [i];
      diagvuu (a, b, ip, jp);
      ip = &s0 [i+2*LOCAL*(TIMESTEP-1)+LOCAL*(WIDTH)];
      jp = &s1 [i+LOCAL*(WIDTH)];
      diagvuu (b, a, ip, jp);
    }
  // }
}

//----------------------------------------------------------------------
#else //REAL2
//----------------------------------------------------------------------


__device__ void diagvu2 (real *ap, real *bp, uint m) {
  // 2*unrolled space-time slice
  int id = threadIdx.x;
  real d0 = bp [0+id];
  real d1 = bp [LOCAL+id];
  for (int i=0; i<2*m*LOCAL; i+=2*LOCAL) {
    real a0 = ap [i+id];
    real a1 = ap [i+LOCAL+id];
    real e0 = kern (a0, a1, d0);
    real e1 = kern (a1, d0, d1);
    bp [i+2*LOCAL+id] = e0;
    bp [i+3*LOCAL+id] = e1;
    d0 = e0;
    d1 = e1;
  }
}

__device__ void diagvuu (real *ap, real *bp, real *ip, real *jp) {
  // u*unrolled in space, 2*in time, space-time slice
  int id = threadIdx.x;
  ap[id] = ip [id];
  ap[LOCAL+id] = ip [LOCAL+id];
  real d[WIDTH+2];
  for (int k=0; k<WIDTH; k++)
    d[k+2] = ip [LOCAL*(k+2)+id];
  for (int i=0; i<2*TIMESTEP*LOCAL; i+=4*LOCAL) {
    d[0] = ap [i+id];
    d[1] = ap [i+LOCAL+id];
    real e[WIDTH+2];
    for (int k=0; k<WIDTH; k++)
      e[k+2] = kern (d[k], d[k+1], d[k+2]);
    bp [i+2*LOCAL+id] = e[WIDTH];
    bp [i+3*LOCAL+id] = e[WIDTH+1];
    e[0] = ap [i+2*LOCAL+id];
    e[1] = ap [i+3*LOCAL+id];
    for (int k=0; k<WIDTH; k++)
      d[k+2] = kern (e[k], e[k+1], e[k+2]);
    bp [i+4*LOCAL+id] = d[WIDTH];
    bp [i+5*LOCAL+id] = d[WIDTH+1];
  }
  for (int k=0; k<WIDTH; k++)
    jp [LOCAL*k+id] = d[k+2];
}


extern "C" __global__ void kernel2 (real *s0, real *s1,
				    real *ga, real *gb) {
  int id = threadIdx.x;
  int k1 = blockIdx.x *LOCAL*2*(TIMESTEP+1);

  real *a = &ga[k1];
  real *b = &gb[k1];
  int i0 = blockIdx.x *LOCAL*GRID_LOCAL*WIDTH;
  int i1 = (blockIdx.x+1) *LOCAL*GRID_LOCAL*WIDTH;
  a [0+id] = s0 [i0+id];
  a [LOCAL+id] = s0 [i0+LOCAL+id];
  for (int i=1; i<TIMESTEP; i++) { // 2* unroll initial
    b [0+id] = s0 [2*LOCAL*i+i0+id];
    b [LOCAL+id] = s0 [2*LOCAL*i+LOCAL+i0+id];
    diagvu2 (a, b, i);
    real *c = a;
    a = b;
    b = c;
  }

  // if (GRID_LOCAL%2 == 1) {
  //   for (int i=i0; i<i1; i+=LOCAL*(WIDTH)) { // u* unroll block
  //     real *ip = &s0 [i+2*LOCAL*(TIMESTEP-1)];
  //     real *jp = &s1 [i];
  //     diagvuu (a, b, ip, jp);
  //     real *c = a;
  //     a = b;
  //     b = c;
  //   }
  // } else {
    for (int i=i0; i<i1; i+=2*LOCAL*(WIDTH)) { // u* unroll block
      real *ip = &s0 [i+2*LOCAL*(TIMESTEP-1)];
      real *jp = &s1 [i];
      diagvuu (a, b, ip, jp);
      ip = &s0 [i+2*LOCAL*(TIMESTEP-1)+LOCAL*(WIDTH)];
      jp = &s1 [i+LOCAL*(WIDTH)];
      diagvuu (b, a, ip, jp);
    }
  // }
}

//----------------------------------------------------------------------
#endif //REAL2
#else // WRP
#ifdef REAL2
//----------------------------------------------------------------------

#ifdef FLOAT
typedef float2 real2;
#else
typedef double2 real2;
#endif

__device__ void diagvu2 (real2 *ap, real2 *bp, uint m) {
  // 2*unrolled space-time slice
  int ida = threadIdx.x + __mul24((int)WRP*(TIMESTEP+1), threadIdx.y);

  real2 dd = bp [0+ida];
  real d0 = dd.x;
  real d1 = dd.y;
  for (int i=0; i<m*WRP; i+=WRP) {
    real2 ad = ap [i+ida];
    real a0 = ad.x;
    real a1 = ad.y;
    real e0 = kern (a0, a1, d0);
    real e1 = kern (a1, d0, d1);
    bp [i+WRP+ida] = (real2){e0, e1};
    d0 = e0;
    d1 = e1;
  }
}

__device__ void diagvuu (real2 *ap, real2 *bp, real *ip, real *jp) {
  // u*unrolled in space, 2*in time, space-time slice
  int id = threadIdx.x + __mul24((int)WRP, threadIdx.y);
  int ida = threadIdx.x + __mul24((int)WRP*(TIMESTEP+1), threadIdx.y);

  ap[ida] = (real2){ip [id], ip [LOCAL+id]};
  real d[WIDTH+2];
  for (int k=0; k<WIDTH; k++)
    d[k+2] = ip [LOCAL*(k+2)+id];
  for (int i=0; i<TIMESTEP*WRP; i+=2*WRP) {
    real2 dd = ap [i+ida];
    d[0] = dd.x;
    d[1] = dd.y;
    real e[WIDTH+2];
    for (int k=0; k<WIDTH; k++)
      e[k+2] = kern (d[k], d[k+1], d[k+2]);
    bp [i+WRP+ida] = (real2){e[WIDTH], e[WIDTH+1]};
    real2 ed = ap [i+WRP+ida];
    e[0] = ed.x;
    e[1] = ed.y;
    for (int k=0; k<WIDTH; k++)
      d[k+2] = kern (e[k], e[k+1], e[k+2]);
    bp [i+2*WRP+ida] = (real2){d[WIDTH], d[WIDTH+1]};
  }
  for (int k=0; k<WIDTH; k++)
    jp [LOCAL*k+id] = d[k+2];
}


extern "C" __global__ void kernel2 (real *s0, real *s1,
				    real *ga, real *gb) {
  int id = threadIdx.x + __mul24((int)WRP, threadIdx.y);
  int ida = threadIdx.x + __mul24((int)WRP*(TIMESTEP+1), threadIdx.y);
  int k1 = blockIdx.x *LOCAL*2*(TIMESTEP+1);

  real2 *a = (real2*)&ga[k1];
  real2 *b = (real2*)&gb[k1];
  int i0 = blockIdx.x *LOCAL*GRID_LOCAL*WIDTH;
  int i1 = (blockIdx.x+1) *LOCAL*GRID_LOCAL*WIDTH;
  a [0+ida] = (real2){s0 [i0+id], s0 [i0+LOCAL+id]};
  for (int i=1; i<TIMESTEP; i++) { // 2* unroll initial
    b [0+ida] = (real2){s0 [2*LOCAL*i+i0+id], s0 [2*LOCAL*i+LOCAL+i0+id]};
    diagvu2 (a, b, i);
    real2 *c = a;
    a = b;
    b = c;
  }

  // if (GRID_LOCAL%2 == 1) {
  //   for (int i=i0; i<i1; i+=LOCAL*(WIDTH)) { // u* unroll block
  //     real *ip = &s0 [i+2*LOCAL*(TIMESTEP-1)];
  //     real *jp = &s1 [i];
  //     diagvuu (a, b, ip, jp);
  //     real2 *c = a;
  //     a = b;
  //     b = c;
  //   }
  // } else {
    for (int i=i0; i<i1; i+=2*LOCAL*(WIDTH)) { // u* unroll block
      real *ip = &s0 [i+2*LOCAL*(TIMESTEP-1)];
      real *jp = &s1 [i];
      diagvuu (a, b, ip, jp);
      ip = &s0 [i+2*LOCAL*(TIMESTEP-1)+LOCAL*(WIDTH)];
      jp = &s1 [i+LOCAL*(WIDTH)];
      diagvuu (b, a, ip, jp);
    }
  // }
}

//----------------------------------------------------------------------
#else //REAL2
//----------------------------------------------------------------------

__device__ void diagvu2 (real *ap, real *bp, uint m) {
  // 2*unrolled space-time slice
  int ida = threadIdx.x + __mul24((int)WRP*2*(TIMESTEP+1), threadIdx.y);

  real d0 = bp [0+ida];
  real d1 = bp [WRP+ida];
  for (int i=0; i<2*m*WRP; i+=2*WRP) {
    real a0 = ap [i+ida];
    real a1 = ap [i+WRP+ida];
    real e0 = kern (a0, a1, d0);
    real e1 = kern (a1, d0, d1);
    bp [i+2*WRP+ida] = e0;
    bp [i+3*WRP+ida] = e1;
    d0 = e0;
    d1 = e1;
  }
}

__device__ void diagvuu (real *ap, real *bp, real *ip, real *jp) {
  // u*unrolled in space, 2*in time, space-time slice
  int id = threadIdx.x + __mul24((int)WRP, threadIdx.y);
  int ida = threadIdx.x + __mul24((int)WRP*2*(TIMESTEP+1), threadIdx.y);

  ap[ida] = ip [id];
  ap[WRP+ida] = ip [LOCAL+id];
  real d[WIDTH+2];
  for (int k=0; k<WIDTH; k++)
    d[k+2] = ip [LOCAL*(k+2)+id];
  for (int i=0; i<2*TIMESTEP*WRP; i+=4*WRP) {
    d[0] = ap [i+ida];
    d[1] = ap [i+WRP+ida];
    real e[WIDTH+2];
    for (int k=0; k<WIDTH; k++)
      e[k+2] = kern (d[k], d[k+1], d[k+2]);
    bp [i+2*WRP+ida] = e[WIDTH];
    bp [i+3*WRP+ida] = e[WIDTH+1];
    e[0] = ap [i+2*WRP+ida];
    e[1] = ap [i+3*WRP+ida];
    for (int k=0; k<WIDTH; k++)
      d[k+2] = kern (e[k], e[k+1], e[k+2]);
    bp [i+4*WRP+ida] = d[WIDTH];
    bp [i+5*WRP+ida] = d[WIDTH+1];
  }
  for (int k=0; k<WIDTH; k++)
    jp [LOCAL*k+id] = d[k+2];
}


extern "C" __global__ void kernel2 (real *s0, real *s1,
				    real *ga, real *gb) {
  int id = threadIdx.x + __mul24((int)WRP, threadIdx.y);
  int ida = threadIdx.x + __mul24((int)WRP*2*(TIMESTEP+1), threadIdx.y);
  int k1 = blockIdx.x *LOCAL*2*(TIMESTEP+1);

  real *a = &ga[k1];
  real *b = &gb[k1];
  int i0 = blockIdx.x *LOCAL*GRID_LOCAL*WIDTH;
  int i1 = (blockIdx.x+1) *LOCAL*GRID_LOCAL*WIDTH;
  a [0+ida] = s0 [i0+id];
  a [WRP+ida] = s0 [i0+LOCAL+id];
  for (int i=1; i<TIMESTEP; i++) { // 2* unroll initial
    b [0+ida] = s0 [2*LOCAL*i+i0+id];
    b [WRP+ida] = s0 [2*LOCAL*i+LOCAL+i0+id];
    diagvu2 (a, b, i);
    real *c = a;
    a = b;
    b = c;
  }

  // if (GRID_LOCAL%2 == 1) {
  //   for (int i=i0; i<i1; i+=LOCAL*(WIDTH)) { // u* unroll block
  //     real *ip = &s0 [i+2*LOCAL*(TIMESTEP-1)];
  //     real *jp = &s1 [i];
  //     diagvuu (a, b, ip, jp);
  //     real *c = a;
  //     a = b;
  //     b = c;
  //   }
  // } else {
    for (int i=i0; i<i1; i+=2*LOCAL*(WIDTH)) { // u* unroll block
      real *ip = &s0 [i+2*LOCAL*(TIMESTEP-1)];
      real *jp = &s1 [i];
      diagvuu (a, b, ip, jp);
      ip = &s0 [i+2*LOCAL*(TIMESTEP-1)+LOCAL*(WIDTH)];
      jp = &s1 [i+LOCAL*(WIDTH)];
      diagvuu (b, a, ip, jp);
    }
  // }
}

//----------------------------------------------------------------------
#endif //REAL2
#endif // WRP


int comp (const void*x, const void*y) {
  double xx = *(double*)x;
  double yy = *(double*)y;
  return xx<yy;
}


real* init_cpu (uint n, uint o, uint local) {
  real *x = (real*)malloc ((n+o) * sizeof (real));
  if (!x) pferror ("malloc");
  for (uint j=0; j<n/local; j++)
    for (uint i=0; i<local; i++) {
      real y = (j+i*(n/local)) / (real)n;
      y = y*y;
      x[j*local+i] = y;
    }
  for (uint j=0; j<o/local; j++)
    for (uint i=0; i<local; i++) {
      x[n+j*local+(i+1)%local] = x[j*local+i];
    }
  return x;
}

void iterate_cpu (real *x, uint n, uint local, uint iter) {
  uint m = n / local;
  for (uint it=0; it<iter; it++) {
    for (uint j=0; j<m-2-it; j++)
      for (uint i=0; i<local; i++) {
	uint l = j*local+i;
	x[l] = kern (x[l], x[l+local], x[l+2*local]);
      }
  }
}

real* read (real * xd, uint n, uint th) {
  real *x = (real*)malloc (n * sizeof (real));
  if (!x) pferror ("malloc");
  pfgpu[th].read (xd, x, 0, n);
  return x;
}

void print (real *x, uint n) {
  for (uint i=0; i<n; i++)
    cout<<x[i]<<" ";
  cout<<"\n";
}

void print (real *x, real *y, uint n) {
  for (uint i=0; i<n; i++)
    cout<<x[i]-y[i]<<" ";
  cout<<"\n";
}

void diff (real *x0, real *x1, uint n) {
  real s1 = 0.f, s2 = 0.f, si = 0.f;
  for (uint i=0; i<n; i++) {
    real y = fabs(x0[i]-x1[i]);
    s1 += y;
    s2 += y*y;
    si = fmaxf(si, y);
  }
  s1 = s1 / n;
  s2 = sqrtf (s2 / n);
  cout<<"error l1="<<s1<<"  error l2="<<s2<<"  error max="<<si<<"\n";
}

void init_gpu (int argc, char *argv[]) {
  uint p = DEV_MAX;
  for (uint i=0; i<p; i++)
    pfgpu[i].init (argc, argv, i);
}

int main (int argc, char *argv[]) {
  const uint local = LOCAL, iter=TIMESTEP, width=WIDTH, grid=GRID_LOCAL, proc=PROC, maxthread=DEV_MAX;
  init_gpu (argc, argv);

  const uint p = maxthread;
  uint n = proc*width*local*grid;
  uint o = local*2*iter;
  if (grid<2*iter) pferror ("overlap too large");
  real *x[maxthread], *y[maxthread], *a[maxthread], *b[maxthread];
  real* x_buf[maxthread];
  for (uint i=0; i<p; i++) {
    x[i] = pfgpu[i].alloc<real> (n+o);
    y[i] = pfgpu[i].alloc<real> (n);
    a[i] = pfgpu[i].alloc<real> (2*(iter+2)*local*proc);
    b[i] = pfgpu[i].alloc<real> (2*(iter+2)*local*proc);
    x_buf[i] = (real*)malloc(sizeof(real)*o);
    if (!x_buf[i]) pferror("malloc");
  }

#ifdef CHECK
#define IT 1
#else
#define IT 1
#endif

#ifdef CHECK
  cout<<"n="<<n<<"\n"<<"o="<<o<<"\n"<<"p="<<p<<"\n"<<"iter="<<iter<<"\n";
  real *xh = init_cpu (n*p, o, local);
#endif // CHECK

  double fl[IT];
  for (uint it=0; it<IT; it++) {

    for (uint i=0; i<p; i++) {
      // cout << "line " << __LINE__ << "\n";
      pfgpu[i].start ();
      kernel0 <<<proc, local>>> (x[i], n, i);
      // cout << "line " << __LINE__ << "\n";
      pfgpu[i].sync ();
    }

    realtime r;
    r.start ();


    if (p>1) {
      for (uint i=0; i<p; i+=2)
       	pfgpu[i].copy (x[(i+1)%p], pfgpu[(i+1)%p], x[i], 0, n, o);

      for (uint i=1; i<p; i+=2)
       	pfgpu[i].copy (x[(i+1)%p], pfgpu[(i+1)%p], x[i], 0, n, o);

      for (uint i=0; i<p; i++) {
	// cout << "line " << __LINE__ << "\n";
	pfgpu[i].start ();
	kernel1 <<<2*iter, local>>> (x[i], n);
      }

      for (uint i=0; i<p; i++) {
	// cout << "line " << __LINE__ << "\n";
	pfgpu[i].sync ();
      }

    }

#ifdef PRINT
    for (uint i=0; i<p; i++) {
      real *xd = read (x[i], n+o, i);
      cout << "init"<<i<<"\n";
      print (xh+n*i, n+o);
      print (xd, n+o);
      free (xd);
    }
#endif // PRINT

    for (uint i=0; i<p; i++) {
      // cout << "line " << __LINE__ << "\n";
      pfgpu[i].start ();
#ifndef WRP
      kernel2 <<<proc, local>>> (x[i], y[i], a[i], b[i]);
#else // WRP
      dim3 p2(proc, 1);
      dim3 l2(WRP, local/WRP);
      kernel2 <<<p2, l2>>> (x[i], y[i], a[i], b[i]);
#endif // WRP
    }

    for (uint i=0; i<p; i++) {
      // cout << "line " << __LINE__ << "\n";
      pfgpu[i].sync ();
    }

    r.stop ();

    // if (p==1) 
    //   fl[it] = pfgpu[0].time();
    // else
    fl[it] = r.elapsed (); // pfgpu[0].time();
    std::cout <<"t_host="<<r.elapsed ()
	      <<"  t_gpu="<<pfgpu[0].time()
	      <<std::endl;

#ifdef CHECK
    iterate_cpu (xh, n*p+o, local, iter);
    for (uint i=0; i<p; i++) {
      real *yd = read (y[i], n, i);
#ifdef PRINT
      cout << "res"<<i<<"\n";
      print (xh+n*i, n);
      print (yd, n);
#endif // PRINT
      diff (xh+n*i, yd, n);
      free (yd);
    }
#endif // CHECK
  }
  qsort (&fl[0], IT, sizeof (fl[0]), comp);
  std::cout<<"flop="<<(p*iter*4.*width*proc*local*grid) / fl[IT/2]<<std::endl;


  for (uint i=0; i<p; i++) {
    free(x_buf[i]);
    pfgpu[i].free (b[i]);
    pfgpu[i].free (a[i]);
    pfgpu[i].free (y[i]);
    pfgpu[i].free (x[i]);
    pfgpu[i].close ();
  }
  hipDeviceReset ();
  return 0;
}
